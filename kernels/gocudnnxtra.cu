
#include <hip/hip_runtime.h>


#define CUDA_GRID_LOOP_X(i, n)                                 \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
         i += blockDim.x * gridDim.x)

#define CUDA_GRID_AXIS_LOOP(i, n, axis)                                 \
    for (int i = blockIdx.axis * blockDim.axis + threadIdx.axis; i < n; \
         i += blockDim.axis * gridDim.axis)

extern "C" __global__ void Transpose(int numthreads,
                                     const float *src,
                                     const int *buf,
                                     const int ndims,
                                     float *dest)
{
    const int *src_strides = buf;
    const int *dest_strides = &buf[ndims];
    const int *perm = &buf[ndims * 2];

    CUDA_GRID_LOOP_X(destIdx, numthreads)
    {
        int srcIdx = 0;
        int t = destIdx;
        for (int i = 0; i < ndims; ++i)
        {
            const int ratio = t / dest_strides[i];
            t -= ratio * dest_strides[i];
            srcIdx += (ratio * src_strides[perm[i]]);
        }
        dest[destIdx] = src[srcIdx];
    }
}

/*SwapEveryOther will swap the batches between 2 tensors. 
 It will be either the even or the odd.
   Both tensors have to be equal in size and dims.
   if even is >0 then it will do the even batches.
   Make sure labels are swapped on host end.
   */
   extern "C" __global__ void SwapEveryOther(
    const int xThreads,
    const int totalbatches,
    float *t1,
    float *t2,
    int even)
{
const int BVol = xThreads;// * yThreads * zThreads;
//const int xVol = yThreads * zThreads;
//const int yVol = zThreads;
    for (int i = 0; i < totalbatches; i++)
    { 
        if (even>0)
        {
            if (i%2==0)
            {
                CUDA_GRID_LOOP_X(xIdx, xThreads)
                {
                   float swapper =  t1[(i*BVol)+(xIdx)];
                   t1[(i*BVol) +xIdx]=t2[(i*BVol)+xIdx];
                   t2[(i*BVol)+xIdx]=swapper;
                    
                }
            }
        }
        else  
        {
            if (i%2==1)
            {
                CUDA_GRID_LOOP_X(xIdx, xThreads)
                {
                   float swapper =  t1[(i*BVol)+(xIdx)];
                   t1[(i*BVol) +xIdx]=t2[(i*BVol)+xIdx];
                   t2[(i*BVol)+xIdx]=swapper;
                    
                }
            }     
            }
        }
    }


//InnerSwapLowerUpper will swap either the upper or lower batches,
//If inverse is >0 then it will swap the first with the last
//If inverse <0 then it will start at the middle instead of the end
extern "C" __global__ void InnerSwapLowerUpper(
    const int xThreads,
    const int totalbatches,
    float *t1,
    const int inverse)
{
const int BVol = xThreads;
  
        if (inverse>0){
            for (int i = 0; i < totalbatches/2; i++)
            { 
            int j =totalbatches-2;
            if (i !=j)
            {
                CUDA_GRID_AXIS_LOOP(xIdx, xThreads, x)
                {
                    const float swapper =  t1[(i*BVol)+(xIdx)];
                    t1[(i*BVol)+(xIdx)]=t1[(j*BVol)+xIdx];
                            t1[(j*BVol)+xIdx]=swapper;
                }
            }
        }
    }
    else
    {
        for (int i = 0; i < totalbatches/2; i++)
        { 
        int j =(totalbatches/2)+i;
            
            if (j<totalbatches)
            {
                CUDA_GRID_AXIS_LOOP(xIdx, xThreads, x)
                {
                    const float swapper =  t1[(i*BVol)+(xIdx)];
                    t1[(i*BVol)+(xIdx)]=t1[(j*BVol)+xIdx];
                            t1[(j*BVol)+xIdx]=swapper;
                }
            }
        }
    }
}
   

//SwapUpperLower will swap either the upper or lower batches
extern "C" __global__ void SwapUpperLower(
    const int xThreads,
    const int totalbatches,
    float *t1,
    float *t2,
    int upper)
{
const int BVol = xThreads;
  
    if (upper>0)
    {
        for (int i = 0; i < totalbatches/2; i++)
        { 
            CUDA_GRID_AXIS_LOOP(xIdx, xThreads, x)
            {
               
                const float swapper =  t1[(i*BVol)+(xIdx)];
                t1[(i*BVol)+(xIdx)]=t2[(i*BVol)+(xIdx)];
                t2[(i*BVol)+(xIdx)]=swapper;
           
            }
        }
    }
    else  
    {
        for (int i =  totalbatches/2; i < totalbatches; i++)
        {           
            CUDA_GRID_AXIS_LOOP(xIdx, xThreads, x)
            {
         
            const float swapper =  t1[(i*BVol)+(xIdx)];
            t1[(i*BVol)+(xIdx)]=t2[(i*BVol)+(xIdx)];
            t2[(i*BVol)+(xIdx)]=swapper;
             
            }      
        }
    }
}
//InnerSwapBatch will swap batch A and B
//Make sure labels are swapped on the host end.
extern "C" __global__ void InnerSwapBatch(
    const int xThreads,
    float *t1,
    const int batchA,
    const int batchB)
{
const int BVol = xThreads; 
if (batchA !=batchB){
    CUDA_GRID_AXIS_LOOP(xIdx, xThreads, x)
    {
        const float swapper =  t1[(batchA*BVol)+(xIdx)];
        t1[(batchA*BVol)+(xIdx)]=t1[(batchB*BVol)+(xIdx)];
        t1[(batchA*BVol)+(xIdx)]=swapper;
    
    }
}    

}
        
//ShapetoBatch4DNHWC Does a stride shape to batch. Make sure values on receiving end are set to zero when s2b is 0
extern "C" __global__ void ShapetoBatch4DNHWC(
    const int xThreads,
    const int yThreads,
    const int zThreads,
    const int hSize,
    const int wSize,
    const int BatchOffset,
    const int ShapeOffset,
    const int N1,
    const int N2,
    const int hstride,
    const int wstride,
    float *shape,
    float *batch,
    int S2B)
{
    int batch0 = N2 * xThreads * yThreads * zThreads;
    int batch1 = xThreads * yThreads * zThreads;
    int batch2 = yThreads * zThreads;
    int batch3 = zThreads;
    for (int i = 0; i < N1; i++)
    {
        for (int j = 0; j < N2; j++)
        {
            CUDA_GRID_AXIS_LOOP(xIdx, xThreads, x)
            {
                CUDA_GRID_AXIS_LOOP(yIdx, yThreads, y)
                {
                    CUDA_GRID_AXIS_LOOP(zIdx, zThreads, z)
                    {

                        int oh = (hstride * i) + xIdx;
                        int ow = (wstride * j) + yIdx;

                        if (S2B > 0)
                        {
                            if (oh < hSize && ow < wSize)
                            {
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] =
                                    shape[ShapeOffset + (oh * hSize * zThreads) + (ow * zThreads) + zIdx];
                            }
                            else
                            {
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] = 0;
                            }
                        }
                        else
                        {
                            shape[ShapeOffset + (oh * hSize * zThreads) + (ow * zThreads) + zIdx] +=
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx];
                        }
                    }
                }
            }
        }
    }
}


//ShapetoBatch4DNCHW Does a stride shape to batch. Make sure values on receiving end are set to zero when s2b is 0


extern "C" __global__ void ShapetoBatch4DNCHW(
    const int xThreads,
    const int yThreads,
    const int zThreads,
    const int hSize,
    const int wSize,
    const int BatchOffset,
    const int ShapeOffset,
    const int N1,
    const int N2,
    const int hstride,
    const int wstride,
    float *shape,
    float *batch,
    int S2B)
{
    int batch0 = N2 * xThreads * yThreads * zThreads;
    int batch1 = xThreads * yThreads * zThreads;
    int batch2 = xThreads * yThreads;
    int batch3 = yThreads;
    for (int i = 0; i < N1; i++)
    {
        for (int j = 0; j < N2; j++)
        {
            CUDA_GRID_AXIS_LOOP(xIdx, xThreads, x)
            {
                CUDA_GRID_AXIS_LOOP(yIdx, yThreads, y)
                {
                    CUDA_GRID_AXIS_LOOP(zIdx, zThreads, z)
                    {

                        int oh = (hstride * i) + yIdx;
                        int ow = (wstride * j) + zIdx;

                        if (S2B > 0)
                        {
                            if (oh < hSize && ow < wSize)
                            {
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] =
                                    shape[ShapeOffset + (xIdx * wSize * hSize) + (oh * wSize) + ow];
                            }
                            else
                            {
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx] = 0;
                            }
                        }
                        else
                        {
                            shape[ShapeOffset + (xIdx * wSize * hSize) + (oh * wSize) + ow] +=
                                batch[BatchOffset + (i * batch0) + (j * batch1) + (xIdx * batch2) + (yIdx * batch3) + zIdx];
                        }
                    }
                }
            }
        }
    }
}

extern "C" __global__ void nearestneighborNHWC(
    const int aligncorners,
    const int threads,
    const float *src,
    const int src_height,
    const int src_width,
    const int channels,
    const int dest_height,
    const int dest_width,
    const float height_scale,
    const float width_scale,
    float *dest)
{
    CUDA_GRID_LOOP_X(i, threads)
    {
        int n = i;
        int c = n % channels;
        n /= channels;
        int dest_x = n % dest_width;
        n /= dest_width;
        int dest_y = n % dest_height;
        n /= dest_height;
        const float *src_data_n = &src[n * channels * src_height * src_width];
        const int src_y = fminf((aligncorners) ? (roundf(dest_y * height_scale))
                                               : (floorf(dest_y * height_scale)),
                                src_height - 1);

        const int src_x = fminf((aligncorners) ? (roundf(dest_x * width_scale))
                                               : (floorf(dest_x * width_scale)),
                                src_width - 1);
        const int idx = (src_y * src_width + src_x) * channels + c;
        dest[i] = src_data_n[idx];
    }
}

//When calling this function it will have to do the stuff indexes on the destination
extern "C" __global__ void nearestneighborv2NCHW(
    const int xThreads,
    const int yThreads,
    const int zThreads,
    const int batches,
    const float *src,
    const int src_height,
    const int src_width,
    // const int dest_height,
    //  const int dest_width,
    const float hratio,
    const float wratio,
    float *dest)
{
    const int dbatchslide = xThreads * yThreads * zThreads;
    const int dchanslide = yThreads * zThreads;
    const int dhslide = zThreads;
    const int schanslide = src_height * src_width;
    const int sbatchslide = schanslide * xThreads;
    for (int i = 1; i < batches; i++)
    {

        CUDA_GRID_AXIS_LOOP(xIdx, xThreads, x)
        {
            CUDA_GRID_AXIS_LOOP(yIdx, yThreads, y)
            {
                CUDA_GRID_AXIS_LOOP(zIdx, zThreads, z)
                {
                    float ph = floorf(yIdx * hratio);
                    float pw = floorf(zIdx * wratio);
                    dest[(i * dbatchslide) + (xIdx * dchanslide) + (yIdx * dhslide) + zIdx] =
                        src[(int)((i * sbatchslide) + (schanslide * xIdx) + (ph * src_height) + pw)];
                }
            }
        }
    }
}
//When calling this function it will have to do the stuff indexes on the destination
extern "C" __global__ void nearestneighborv2NCHWAddGradient(
    const int xThreads,
    const int yThreads,
    const int zThreads,
    const int batches,
    const float *src,
    const int src_height,
    const int src_width,
    // const int dest_height,
    //  const int dest_width,
    const float hratio,
    const float wratio,
    float *dest)
{
    const int dbatchslide = xThreads * yThreads * zThreads;
    const int dchanslide = yThreads * zThreads;
    const int dhslide = zThreads;
    const int schanslide = src_height * src_width;
    const int sbatchslide = schanslide * xThreads;
    for (int i = 1; i < batches; i++)
    {

        CUDA_GRID_AXIS_LOOP(xIdx, xThreads, x)
        {
            CUDA_GRID_AXIS_LOOP(yIdx, yThreads, y)
            {
                CUDA_GRID_AXIS_LOOP(zIdx, zThreads, z)
                {
                    float ph = floorf(yIdx * hratio);
                    float pw = floorf(zIdx * wratio);
                    dest[(i * dbatchslide) + (xIdx * dchanslide) + (yIdx * dhslide) + zIdx] +=
                        src[(int)((i * sbatchslide) + (schanslide * xIdx) + (ph * src_height) + pw)];
                }
            }
        }
    }
}
extern "C" __global__ void nearestneighborNCHW(
    const int aligncorners,
    const int threads,
    const float *src,
    const int src_height,
    const int src_width,
    const int channels,
    const int dest_height,
    const int dest_width,
    const float height_scale,
    const float width_scale,
    float *dest)
{
    CUDA_GRID_LOOP_X(i, threads)
    {
        int n = i;
        int dest_x = n % dest_width;
        n /= dest_width;
        int dest_y = n % dest_height;
        n /= dest_height;
        int c = n % channels;
        n /= channels;
        const float *src_data_n = &src[n * channels * src_height * src_width];
        const int src_y = fminf((aligncorners) ? (roundf(dest_y * height_scale))
                                               : (floorf(dest_y * height_scale)),
                                src_height - 1);

        const int src_x = fminf((aligncorners) ? (roundf(dest_x * width_scale))
                                               : (floorf(dest_x * width_scale)),
                                src_width - 1);
        const int idx = (c * src_height * src_width) + (src_y * src_width) + src_x;
        dest[i] = src_data_n[idx];
    }
}
extern "C" __global__ void nearestneighborNCHWBack(
    const int aligncorners,
    const int threads,
    float *src,
    const int src_height,
    const int src_width,
    const int channels,
    const int dest_height,
    const int dest_width,
    const float height_scale,
    const float width_scale,
    float *dest)
{
    CUDA_GRID_LOOP_X(i, threads)
    {
        int n = i;
        int src_x = n % src_width;
        n /= src_width;
        int src_y = n % src_height;
        n /= src_height;
        int c = n % channels;
        n /= channels;
        float *src_data_n = &src[n * channels * src_height * src_width];
        const int dest_y = fminf((aligncorners) ? (roundf(src_y * height_scale))
                                                : (floorf(src_y * height_scale)),
                                 dest_height - 1);

        const int dest_x = fminf((aligncorners) ? (roundf(src_x * width_scale))
                                                : (floorf(src_x * width_scale)),
                                 dest_width - 1);
        const int idx = (c * dest_width * dest_height) + (dest_y * dest_width) + dest_x;
        atomicAdd(&src_data_n[idx], dest[i]);
    }
}
extern "C" __global__ void nearestneighborNHWCBack(
    const int aligncorners,
    const int threads,
    float *src,
    const int src_height,
    const int src_width,
    const int channels,
    const int dest_height,
    const int dest_width,
    const float height_scale,
    const float width_scale,
    float *dest)
{
    CUDA_GRID_LOOP_X(i, threads)
    {
        int n = i;
        int c = n % channels;
        n /= channels;
        int src_x = n % src_width;
        n /= src_width;
        int src_y = n % src_height;
        n /= src_height;
        float *src_data_n = &src[n * channels * src_height * src_width];
        const int dest_y = fminf((aligncorners) ? (roundf(src_y * height_scale))
                                                : (floorf(src_y * height_scale)),
                                 dest_height - 1);

        const int dest_x = fminf((aligncorners) ? (roundf(src_x * width_scale))
                                                : (floorf(src_x * width_scale)),
                                 dest_width - 1);
        const int idx = (dest_y * dest_width + dest_x) * channels + c;
        atomicAdd(&src_data_n[idx], dest[i]);
    }
}

extern "C" __global__ void adagradfloat(const int length,
                                        float *weights,   //weights input and output
                                        float *dw,        //input and will have to set to zero
                                        float *gsum,      //storage
                                        const float rate, //input
                                        const float eps)
{ //input
    CUDA_GRID_LOOP_X(cell, length)
    {

        int holder = gsum[cell];
        gsum[cell] = holder + (dw[cell] * dw[cell]);
        weights[cell] = -(rate * dw[cell]) / (sqrtf(gsum[cell]) + eps);
        dw[cell] = 0.0;
    }
}

extern "C" __global__ void adamfloat(const int length,
                                     float *w,
                                     float *gsum,
                                     float *xsum,
                                     float *dw,
                                     const float rate,
                                     const float beta1,
                                     const float beta2,
                                     const float eps,
                                     const float counter)
{

    CUDA_GRID_LOOP_X(i, length)
    {

        gsum[i] = (beta1 * gsum[i]) + ((1.0 - beta1) * dw[i]);
        float gsumt = gsum[i] / (1.0 - powf(beta1, counter));
        xsum[i] = (beta2 * xsum[i]) + ((1.0 - beta2) * (dw[i] * dw[i]));
        float xsumt = xsum[i] / (1.0 - powf(beta2, counter));
        w[i] += -(rate * gsumt) / (sqrtf(xsumt) + eps);
        dw[i] = 0.0;
    }
}

extern "C" __global__ void adadeltafloat(const int length,
                                         float *weights,   //weights input and output
                                         float *gsum,      //storage
                                         float *xsum,      //storage
                                         float *dw,        //input and will have to set to zero
                                         const float rate, //input
                                         const float eps)
{

    CUDA_GRID_LOOP_X(cell, length)
    {

        gsum[cell] = gsum[cell] + (dw[cell] * dw[cell]);
        weights[cell] = -(rate * dw[cell]) / (sqrtf(gsum[cell]) + eps);
        dw[cell] = 0.0;
    }
}

//This is paired with the host
extern "C" __global__ void Segment1stDim(const int start_index, const float *src, float *dst, const int size)
{
    int i = (blockIdx.y * gridDim.x * blockDim.x) + (blockIdx.x * blockDim.x) + threadIdx.x;
    int start_location = start_index * size;
    if (i < size)
    {
        dst[i] = src[start_location + i];
    }
}

extern "C" __global__ void l1l2regularizationfloat(
    const int length,
    float *dw,          //input and output
    const float *w,     //input needs to ba an array
    float *l1,          //output set to zero
    float *l2,          //output set to zero
    const float batch,  // should be an int but just send it as a float
    const float decay1, //input
    const float decay2)
{ //input

    CUDA_GRID_LOOP_X(i, length)
    {

        atomicAdd(l1, abs(w[i]) * decay1);
        atomicAdd(l2, (w[i] * w[i] * decay2) / 2.0);
        const float gradl1 = decay1 * (w[i] > 0 ? 1 : -1);
        const float gradl2 = w[i] * decay2;
        dw[i] = (dw[i] + gradl2 + gradl1) / batch;
    }
}

/*
extern "C" __global__
void l1l2regularizationfloat(
    const int length,
    float *dw, //input and output
    float *w,  //input needs to ba an array
    float *l1, //output set to zero
    float *l2, //output set to zero
    const float batch, // should be an int but just send it as a float
    const float decay1, //input
    const float decay2){ //input
if (decay1 ==0 && decay2==0){
CUDA_GRID_LOOP_X(i,length){ 
        dw[i]/=batch;
}
}else if (decay1==0 && decay2!=0){
CUDA_GRID_LOOP_X(i,length){ 
        atomicAdd(l2,(w[i]*w[i]*decay2)/2.0);
        dw[i]= (dw[i] + w[i]*decay2)/batch;
}
}else if(decay2 == 0 && decay1 !=0){
float decay = decay1;
CUDA_GRID_LOOP_X(i,length){
        if (w[i]<0){
             decay=-decay1;
        }else{
            decay=decay1;
        }
            atomicAdd(l1,w[i]*decay);
            dw[i]= (dw[i] +decay1)/batch;
}
}else if (decay2 !=0 && decay1 !=0) {
float decay = decay1;
CUDA_GRID_LOOP_X(i,length){

        if (w[i]<0){
            decay=-decay1;
        }else{
            decay=decay1;
        }

        atomicAdd(l1,w[i]*decay); 
        atomicAdd(l2,(w[i]*w[i]*decay2)/2.0);
        dw[i]= (dw[i] + (w[i]*decay2) +decay1)/batch;
}
}

}

*/
extern "C" __global__ void AdvanceThreshRandomReluForward(const int length,
                                                          const int batchs,
                                                          const float alpha,
                                                          const float beta,
                                                          const float *x,
                                                          float *y,
                                                          const float *coefs,
                                                          const float *threshhold,
                                                          const int PropNan)
{
    for (int i = 0; i < batchs; i++)
    {
        int stride = length * i;
        CUDA_GRID_LOOP_X(j, length)
        {
            /*
            if (x[stride+j]>alpha){
                y[stride+j]= x[stride+j]*threshhold[j];
         
           
        }else if (x[stride+j]<beta){
          
                y[stride+j]= x[stride+j]*coefs[j];
       
        }else{
            y[stride+j]= x[stride+j];
        }   
        */

            if (x[stride + j] > threshhold[j])
            {

                y[stride + j] = x[stride + j];
            }
            else
            {

                y[stride + j] = x[stride + j] * coefs[j];
            }
        }
        __syncthreads();
    }
}
extern "C" __global__ void AdvanceThreshRandomReluBackward(const int length,
                                                           const int batchs,
                                                           const float alpha,
                                                           const float beta,
                                                           const float *x,
                                                           float *dx,
                                                           const float *dy,
                                                           const float *coefs,
                                                           const float *threshhold,
                                                           const int PropNan)
{

    for (int i = 0; i < batchs; i++)
    {
        int stride = length * i;

        CUDA_GRID_LOOP_X(j, length)
        {
            /*
        if (x[stride+j]>alpha){
            dx[stride+j]= dy[stride+j]*threshhold[j];
     
       
    }else if (x[stride+j]<beta){
      
            dx[stride+j]= dy[stride+j]*coefs[j];
   
    }else{
        dx[stride+j]= dy[stride+j];
    }   
        
    */

            if (x[stride + j] > threshhold[j])
            {
                dx[stride + j] = dy[stride + j];
            }
            else
            {

                dx[stride + j] = dy[stride + j] * coefs[j];
            }
        }
        __syncthreads();
    }
}

extern "C" __global__ void forwardParametricfloatchannel(const int tx,
                                                         const int ty,
                                                         const int tz,
                                                         const int batchindex,
                                                         const float alpha,
                                                         const float beta,
                                                         const float *xx,
                                                         float *yy,
                                                         const float *coefs,
                                                         const int NHWC,
                                                         const int PropNan)
{

    const int stride = tx * ty * tz * batchindex;
    const int ofx = ty * tz;
    const int ofy = tz;
    if (NHWC > 0)
    {
        CUDA_GRID_AXIS_LOOP(i, tx, x)
        {

            CUDA_GRID_AXIS_LOOP(j, ty, y)
            {

                CUDA_GRID_AXIS_LOOP(k, tz, z)
                {
                    int xyindex = stride + (i * ofx) + (j * ofy) + k;

                    float value = (alpha * xx[xyindex] * (xx[xyindex] > 0)) + (alpha * xx[xyindex] * (xx[xyindex] <= 0) * coefs[k]) + (beta * yy[xyindex]);
                    if (PropNan > 0)
                    {
                        yy[xyindex] = value;
                    }
                    else
                    {
                        yy[xyindex] = value * (!(isnan(value) == 0));
                    }
                }
            }
        }
    }
    else
    {
        CUDA_GRID_AXIS_LOOP(i, tx, x)
        {

            CUDA_GRID_AXIS_LOOP(j, ty, y)
            {

                CUDA_GRID_AXIS_LOOP(k, tz, z)
                {
                    int xyindex = stride + (i * ofx) + (j * ofy) + k;

                    float value = (alpha * xx[xyindex] * (xx[xyindex] > 0)) + (alpha * xx[xyindex] * (xx[xyindex] <= 0) * coefs[i]) + (beta * yy[xyindex]);
                    if (PropNan > 0)
                    {
                        yy[xyindex] = value;
                    }
                    else
                    {
                        yy[xyindex] = value * (!(isnan(value) == 0));
                    }
                }
            }
        }
    }
}
//backwardParametricfloat does the backprop of the parametric float

//f(x) = beta*Max(0,x)+alpha*Min(0,x)
extern "C" __global__ void backwardParametricfloatchannel(const int tx,
                                                          const int ty,
                                                          const int tz,
                                                          const int batchindex,
                                                          const float alpha,
                                                          const float beta,
                                                          const float *xx,
                                                          float *dx,
                                                          const float *dy,
                                                          const float *alphas,
                                                          float *dalphas,
                                                          const int NHWC,
                                                          const int PropNan)
{
    int stride = tx * ty * tz * batchindex;
    int ofx = ty * tz;
    int ofy = tz;
    if (NHWC > 0.0)
    {

        CUDA_GRID_AXIS_LOOP(i, tx, x)
        {
            CUDA_GRID_AXIS_LOOP(j, ty, y)
            {
                CUDA_GRID_AXIS_LOOP(k, tz, z)
                {
                    int xyindex = stride + (i * ofx) + (j * ofy) + k;
                    dx[xyindex] = (alpha * dy[xyindex] * (xx[xyindex] > 0)) + ((xx[xyindex] <= 0) * alphas[k] * alpha) + (beta * dx[xyindex]);
                    float value = dy[xyindex] * xx[xyindex] * (xx[xyindex] <= 0);
                    atomicAdd(&dalphas[k], value);
                }
            }
        }
    }
    else
    {
        CUDA_GRID_AXIS_LOOP(i, tx, x)
        {
            CUDA_GRID_AXIS_LOOP(j, ty, y)
            {
                CUDA_GRID_AXIS_LOOP(k, tz, z)
                {
                    int xyindex = stride + (i * ofx) + (j * ofy) + k;
                    dx[xyindex] = (alpha * dy[xyindex] * (xx[xyindex] > 0)) + ((xx[xyindex] <= 0) * alphas[i] * alpha) + (beta * dx[xyindex]);
                    float value = dy[xyindex] * xx[xyindex] * (xx[xyindex] <= 0);
                    atomicAdd(&dalphas[i], value);
                }
            }
        }
    }
}

extern "C" __global__ void forwardleakyfloat(const int length,
                                             const float alpha,
                                             const float beta,
                                             const float *x,
                                             float *y,
                                             const float coef,
                                             const int PropNan)
{
    CUDA_GRID_LOOP_X(i, length)
    {
        if (x[i] > 0.0)
        {
            y[i] = x[i];
        }
        else
        {
            y[i] = x[i] * coef;
        }
    }
}
extern "C" __global__ void backwardleakyfloat(const int length,
                                              const float alpha,
                                              const float beta,
                                              const float *x,
                                              float *dx,
                                              const float *dy,
                                              const float coef,
                                              const int PropNan)
{

    CUDA_GRID_LOOP_X(i, length)
    {

        if (x[i] > 0.0)
        {

            dx[i] = dy[i];
        }
        else
        {

            dx[i] = dy[i] * coef;
        }
    }
}
/*
extern "C" __global__
void forwardleakyfloat(const int length,
                       const float alpha,
                       const float beta,
                       const float *x,
                             float *y,
                       const float coef,
                       const int PropNan){
    CUDA_GRID_LOOP_X(i,length){
        if (x[i]>0.0){
            float value = x[i]*alpha;
            float value2 = y[i]*beta;
            y[i]=value +value2;
           
        }else{
            float value = x[i]*alpha*coef;
            float value2 = y[i]*beta;
            y[i]=value+value2;
         
        }
    }  
    
}
*/
/*
extern "C" __global__
void forwardleakyfloat(const int length,
                       const float alpha,
                       const float beta,
                       const float *x,
                             float *y,
                       const float coef,
                       const int PropNan){
    CUDA_GRID_LOOP_X(i,length){
        if (x[i]>0.0){
            float value = (alpha+x[i])+(beta*y[i]);
            if (PropNan>0){
            y[i]=value;
            }else{
            y[i]=value*(!(isnan(value)==0));
            }
        }else{
            float value = (x[i]*coef*alpha)+(beta*y[i]);
            if (PropNan>0){
            y[i]=value;
            }else{
            y[i]=value*(!(isnan(value)==0));
            }
        }
    }  
    
}

 */
/*

extern "C" __global__
void backwardleakyfloat(const int length,
                        const float alpha,
                        const float beta,
                        const float *x, 
                              float *dx,
                        const float *dy, 
                        const float coef,
                        const int PropNan){

CUDA_GRID_LOOP_X(i,length){

    if (x[i]>0.0){
        float value = dy[i]*alpha;
        float value2 = dx[i]*beta;
        dx[i]=value+value2;
    }else{
        float value = dy[i]*alpha*coef;
        float value2 = dx[i]*beta;
        dx[i]=value+value2;
    }
    
}
}  
*/
/*
extern "C" __global__
void backwardleakyfloat(const int length,
                        const float alpha,
                        const float beta,
                        const float *x, 
                              float *dx,
                        const float *dy, 
                        const float coef,
                        const int PropNan){

CUDA_GRID_LOOP_X(i,length){

    if (x[i]>0.0){
    float value=(dy[i]*alpha)+(beta*dx[i]);
    if (PropNan>0){
        dx[i]=value;
        }else{
    dx[i]=value*(!(isnan(value)==0));
        }
    }else{
        float value= (dy[i]*coef*alpha)+(beta*dx[i]);
        if (PropNan>0){
            dx[i]=value;
            }else{
        dx[i]=value*(!(isnan(value)==0));
            }
    }
    
}

}  
*/
extern "C" __global__ void MSELoss(const int length, float *errors, const float *target, const float *networkout, float *loss)
{

    CUDA_GRID_LOOP_X(i, length)
    {
        const float y = networkout[i] - target[i];
        errors[i] = y;
        atomicAdd(loss, (y * y) / 2);
    }
}