
#include <hip/hip_runtime.h>


#define CUDA_GRID_LOOP_X(i,n)\
for (int i=blockIdx.x *blockDim.x+threadIdx.x;i<n;\
    i +=blockDim.x*gridDim.x)\

#define BLOCK4D_DIMS 2

extern "C" __global__ 
void Transpose(int numthreads,
              const float *src,
              const int *buf,
              const int ndims,
              float *dest){
const int* src_strides=buf;
const int* dest_strides=buf+ndims;
const int* perm=buf+ndims*2;

CUDA_GRID_LOOP_X(destIdx,numthreads){
    int srcIdx=0;
    int t=destIdx;
         for (int i=0;i<ndims;++i){
             const int ratio=t/dest_strides[i];
             t-= ratio * dest_strides[i];
             srcIdx+=ratio *src_strides[perm[i]];
         }
         dest[destIdx]=src[srcIdx];
    }
}
extern "C" __global__
void ShapetoBatch4D(          int BatchShape[BLOCK4D_DIMS+2],
                              int ShapeShape[BLOCK4D_DIMS],
                              int BlockShape[BLOCK4D_DIMS],
                             const int numthrds,
                              const int ShapeBatch,
                             const bool B2S,
                              float *BatchedMem,
                              float *ShapeMem){
CUDA_GRID_LOOP_X(batchIdx,numthrds){
int batchIdxRemainder = batchIdx;
int  batchPos[BLOCK4D_DIMS+2];
for (int dim= BLOCK4D_DIMS+1;dim>=1;--dim){
    batchPos[dim]=batchIdxRemainder%BatchShape[dim];
    batchIdxRemainder/=BatchShape[dim];
}
batchPos[0]=batchIdxRemainder;
int blockIdxRemainder = batchPos[0]/ShapeBatch;
int shapeIdx = batchPos[BLOCK4D_DIMS+1];
int shapeStride =ShapeShape[BLOCK4D_DIMS+1];

const int spaceBatchPos=batchPos[0]%ShapeBatch;
for (int block_Dim=BLOCK4D_DIMS-1;block_Dim>=0;--block_Dim){
    int offset=blockIdxRemainder;
if( block_Dim>0){
    offset %=BlockShape[block_Dim];
}
int shapePos=batchPos[block_Dim+1]*BlockShape[block_Dim]+offset;
if (shapePos>=ShapeShape[block_Dim]){
    if(B2S==false){
        BatchedMem[batchIdx]=0;
    }
    break;
}
shapeIdx+=shapeStride*shapePos;
shapeStride*=ShapeShape[block_Dim];
if (block_Dim==0){
    shapeIdx+=shapeStride*spaceBatchPos;
    if (B2S==false){
        BatchedMem[batchIdx]= ShapeMem[batchIdx];
    }else{
        ShapeMem[shapeIdx]= BatchedMem[batchIdx];
    }
}
blockIdxRemainder/=BlockShape[block_Dim];
}
}


}

extern "C" __global__
void adagradfloat(const int length,
                  float *weights, //weights input and output
                  float *dw, //input and will have to set to zero
                  float *gsum, //storage
                  const float rate, //input
                  const float eps){ //input
                
 
    int section = blockIdx.x;
    int index = threadIdx.x;
    int cell = section*blockDim.x +index;
    if (cell<length){
        int holder = gsum[cell];
        gsum[cell]= holder +(dw[cell]*dw[cell]);
        weights[cell]= -(rate*dw[cell])/(sqrtf(gsum[cell])+eps);
        dw[cell]=0.0;
    }  

}


extern "C" __global__
void adamfloat(const int length,
               float *w,
               float *gsum,
               float *xsum,
               float *dw,
               const float rate,
               const float beta1,
               const float beta2,
               const float eps,
               const float counter){

    
    int i = (blockIdx.y*gridDim.x*blockDim.x) + (blockIdx.x*blockDim.x) +  threadIdx.x;

if (i<length){
     gsum[i]=(beta1*gsum[i]) +((1.0-beta1)*dw[i]);
    float gsumt = gsum[i]/(1.0- powf(beta1,counter));
     xsum[i]= (beta2*xsum[i])+((1.0 -beta2)*(dw[i]*dw[i]));
    float xsumt = xsum[i]/(1.0 - powf(beta2,counter));
    w[i] += -(rate*gsumt)/(sqrtf(xsumt)+eps);  
    dw[i]=0.0;

}
    
}


extern "C" __global__
void adadeltafloat( const int length,
                    float *weights, //weights input and output
                    float *gsum, //storage
                    float *xsum, //storage
                    float *dw, //input and will have to set to zero
                    const float rate, //input
                    const float eps){



            int section = blockIdx.x;
            int index = threadIdx.x;
            int cell = section*blockDim.x +index;
if(cell<length){
    gsum[cell]= gsum[cell]+(dw[cell]*dw[cell]);
    weights[cell]= -(rate*dw[cell])/(sqrtf(gsum[cell])+eps);
    dw[cell]=0.0;


}

}

extern "C" __global__
void l1regularizationfloat(const int length,
                           float *dw, //input and output
                           float *w,  //input
                           float *l1, //output
                           float *l2, //output
                           const float batch, // should be an int but just send it as a float
                           const float decay1,
                           const float decay2){

        int section = blockIdx.x;
        int index = threadIdx.x;
        int cell = section*blockDim.x+index;
        float decay = decay1;
        if (cell<length){
            if (dw[cell]<0){
                decay=-decay;
            }
            atomicAdd(l1,w[cell]*decay);
            dw[cell]= (dw[cell]/batch) +decay;


       }

    
}

//This is paired with the host
extern "C" __global__
void Segment1stDim(const int start_index, const float *src,float *dst ,const int size){
    int i=  (blockIdx.y*gridDim.x*blockDim.x) +(blockIdx.x*blockDim.x) + threadIdx.x;
    int start_location=start_index*size;
    if (i<size){
        dst[i]=src[start_location+i];
    }
}



extern "C" __global__
void l2regularizationfloat(
    const int length,
    float *dw, //input and output
    float *w,  //input
    float *l1, //output
    float *l2, //output
    const float batch, // should be an int but just send it as a float
    const float decay1,
    const float decay2){
        int i=  (blockIdx.y*gridDim.x*blockDim.x) +(blockIdx.x*blockDim.x) + threadIdx.x;
    
    if (i<length){
    atomicAdd(l2,(w[i]*w[i]*decay2)/2.0);
    dw[i]= (dw[i]/batch) + w[i]*decay2;
    }
 
}  
extern "C" __global__
void batchregfloat(
    const int length,
    float *dw, //input and output
    const float batch) {// should be an int but just send it as a float
        int i=  (blockIdx.y*gridDim.x*blockDim.x) +(blockIdx.x*blockDim.x) + threadIdx.x;
    if (i<length){
   
    dw[i]/=batch;
    }
 
}  
extern "C" __global__
void l1l2regularizationfloat(
    const int length,
    float *dw, //input and output
    float *w,  //input needs to ba an array
   // int values, //number of values
    float *l1, //output set to zero
    float *l2, //output set to zero
    const float batch, // should be an int but just send it as a float
    const float decay1, //input
    const float decay2){ //input
 
int i=  (blockIdx.y*gridDim.x*blockDim.x) +(blockIdx.x*blockDim.x) + threadIdx.x;
   int decay =decay1 ;
    if (i<length){
        
        if (dw[i]<0){
            decay=-decay;
        }

        atomicAdd(l1,w[i]*decay); 
        atomicAdd(l2,(w[i]*w[i]*decay2)/2.0);
        dw[i]= (dw[i]/batch) + (w[i]*decay2) +decay1;
     }

}  


extern "C" __global__
void forwardParametricfloat(const int length, const int alphalength ,float *x,float *y,  float *alpha){
   int xsize = gridDim.x*blockDim.x;
   int i= blockIdx.x*blockDim.x+threadIdx.x;
   int j = xsize*blockIdx.y+i;

  
if (j<length){
    if (i<alphalength){
        if (x[j]>0.0){
            y[j]=x[j];
        }else{
            y[j]=alpha[i]*x[j];
        }


    }
    
    
}

}

//NHCW, NCWH only matters on the batch channel so for this to work alpha and dalpha are going to have to be the size of 
// HCW.  
extern "C" __global__  
void backwardParametricfloat(const int length, const int alphalength ,float *x, float *dx,float *dy,  float *alpha, float *dalpha){

    int xsize = gridDim.x*blockDim.x;
    int i= blockIdx.x*blockDim.x+threadIdx.x;
    int j = xsize*blockIdx.y+i;
 
if (j<length){
    if (i<alphalength){
    if (x[j]>0.0){
        dx[j]=dy[j];
    }else{
        dx[j]=alpha[i]*dy[j];
        atomicAdd(&dalpha[i],x[j]*dy[j]);
  
    }   
 
}
}
}
extern "C" __global__
void forwardleakyfloat(const int length,float *x,float *y, const float alpha){

int i=  (blockIdx.y*gridDim.x*blockDim.x) +(blockIdx.x*blockDim.x) + threadIdx.x;
if (i<length){
    if (x[i]>0.0){
        y[i]=x[i];
    }else{
        y[i]=alpha*x[i];
    }
    
}

}   

/*
extern "C" __global__
void concatforwardleakyfloatNCHW(const int length, const int batch, const int xAlength, const int xBlength, const int ylength, float *xA, float xB, float *y, const float alpha){

int i=  (blockIdx.y*gridDim.x*blockDim.x) +(blockIdx.x*blockDim.x) + threadIdx.x;
if (i<xAlength){
    if (xA[i*batch]>0.0){
        y[i]=xA[i*batch];
    }else{
        y[i*batch*(ylength)]=alpha*xA[i*batch];
    }
    
}
if (i<xBlength){
    if (xA[i*batch]>0.0){
        y[i]=xB[i*batch];
    }else{
        y[i*batch*(ylength+xAlength)]=alpha*xB[i*batch];
    }
    
}
}   
*/

extern "C" __global__
void backwardleakyfloat(const int length,float *x, float *dx,float *dy, const float alpha){
int i=  (blockIdx.y*gridDim.x*blockDim.x) +(blockIdx.x*blockDim.x) + threadIdx.x;
if (i<length){
    if (x[i]>0.0){
        dx[i]=dy[i];
    }else{
        dx[i]=alpha*dy[i];
    }
    
}

}  
// Doesn't Work and probably wont be used

/*  
extern "C" __global__
void NHWCSegmentedFrom1NHWC(  const int ChannelIndex,
                              const int ChannelLength,
                              const int OriginalXSize,
                              const int OriginalYSize,
                              float *oMem,
                              float *nMem){

              
int y = (blockIdx.y*blockDim.y +threadIdx.y);//Where y is in the memory    (y is the row)                                                  
int x=  (blockIdx.x*blockDim.x +threadIdx.x);
int ylength= blockDim.y*gridDim.y;
int xlength =(blockDim.x*gridDim.x);
int stridex=gridDim.x*xlength;
int stridey=gridDim.y*ylength;
int OriginalY=stridey+y;
int OriginalX=stridex+x;
__shared__ float *SharedMem;
            if (y<ylength&&x<xlength){
                if  (OriginalX<OriginalXSize && OriginalY<OriginalYSize ){
              SharedMem[x*ylength+y] =  oMem[(ChannelIndex*OriginalXSize*OriginalYSize)+(OriginalX*OriginalYSize)+OriginalY];  
            }else{
                SharedMem[x*ylength+y] =0.0;
            }

                
            }
            __syncthreads();
            nMem[(x*gridDim.y*ChannelLength*ylength*xlength)+(y*ChannelLength*ylength*xlength)+(ChannelIndex*ylength*xlength)+(x*ylength)+y]=  SharedMem[x*ylength+y] ;
          
        } 
        
*/        
/*
extern "C" __global__
void NHWCSegmentedFrom1NHWC(  const int N1index,
                              const int N2index,
                              const int N2length,
                              const int ChannelIndex,
                              const int ChannelLength,
                              const int OriginalXSize,
                              const int OriginalYSize,
                              float *oMem,
                              float *nMem){

              
int y = (blockIdx.y*blockDim.y +threadIdx.y);//Where y is in the memory    (y is the row)                                                  
int x=  (blockIdx.x*blockDim.x +threadIdx.x);
int ylength= blockDim.y*gridDim.y;
int xlength =(blockDim.x*gridDim.x);
int stridex=N1index*xlength;
int stridey=N2index*ylength;
int OriginalY=stridey+y;
int OriginalX=stridex+x;
__shared__ float *SharedMem;
            if (y<ylength&&x<xlength){
                if  (OriginalX<OriginalXSize && OriginalY<OriginalYSize ){
              SharedMem[x*ylength*y] =  oMem[(ChannelIndex*OriginalXSize*OriginalYSize)+(OriginalX*OriginalYSize)+OriginalY];  
            }else{
                SharedMem[x*ylength*y] =0.0;
            }

                
            }
            __syncthreads();
            nMem[(x*N2length*ChannelLength*ylength*xlength)+(y*ChannelLength*ylength*xlength)+(ChannelIndex*ylength*xlength)+(x*ylength)+y]=  SharedMem[x*ylength*y] ;
          
        } 
       



  */     