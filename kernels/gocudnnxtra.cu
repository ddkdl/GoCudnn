
#include <hip/hip_runtime.h>
extern "C" __global__
void adagradfloat(const int length,
                  float *weights, //weights input and output
                  float *dw, //input and will have to set to zero
                  float *gsum, //storage
                  const float rate, //input
                  const float eps){ //input
                
 
    int section = blockIdx.x;
    int index = threadIdx.x;
    int cell = section*blockDim.x +index;
    if (cell<length){
        int holder = gsum[cell];
        gsum[cell]= holder +(dw[cell]*dw[cell]);
        weights[cell]= -(rate*dw[cell])/(sqrtf(gsum[cell])+eps);
        dw[cell]=0.0;
    }  

}


extern "C" __global__
void adamfloat(const int length,
               float *w,
               float *gsum,
               float *xsum,
               float *dw,
               const float rate,
               const float beta1,
               const float beta2,
               const float eps,
               const float counter){

    
    int i = (blockIdx.y*gridDim.x*blockDim.x) + (blockIdx.x*blockDim.x) +  threadIdx.x;

if (i<length){
     gsum[i]=(beta1*gsum[i]) +((1.0-beta1)*dw[i]);
    float gsumt = gsum[i]/(1.0- powf(beta1,counter));
     xsum[i]= (beta2*xsum[i])+((1.0 -beta2)*(dw[i]*dw[i]));
    float xsumt = xsum[i]/(1.0 - powf(beta2,counter));
    w[i] += -(rate*gsumt)/(sqrtf(xsumt)+eps);  
    dw[i]=0.0;

}
    
}


extern "C" __global__
void adadeltafloat( const int length,
                    float *weights, //weights input and output
                    float *gsum, //storage
                    float *xsum, //storage
                    float *dw, //input and will have to set to zero
                    const float rate, //input
                    const float eps){



            int section = blockIdx.x;
            int index = threadIdx.x;
            int cell = section*blockDim.x +index;
if(cell<length){
    gsum[cell]= gsum[cell]+(dw[cell]*dw[cell]);
    weights[cell]= -(rate*dw[cell])/(sqrtf(gsum[cell])+eps);
    dw[cell]=0.0;


}

}

extern "C" __global__
void l1regularizationfloat(const int length,
                           float *dw, //input and output
                           float *w,  //input
                           float *l1, //output
                           float *l2, //output
                           const float batch, // should be an int but just send it as a float
                           const float decay1,
                           const float decay2){

        int section = blockIdx.x;
        int index = threadIdx.x;
        int cell = section*blockDim.x+index;
        float decay = decay1;
        if (cell<length){
            if (dw[cell]<0){
                decay=-decay;
            }
            atomicAdd(l1,w[cell]*decay);
            dw[cell]= (dw[cell]/batch) +decay;


       }

    
}  
extern "C" __global__
void NCHWsegmentfrom1CHWfloat(const int BatchIndex,
                              const int MetaBlockIdxX,
                              const int MetaBlockIdxY,
                              const int MetaBlockIdxZ,
                              const int MetaGridDimX,
                              const int MetaGridDimY,
                              const int MetaGridDimZ,
                              const int OriginalTotalVolume,
                              float *oMem,
                              float *nMem){


int MetaID = MetaBlockIdxX +(MetaBlockIdxY*MetaGridDimX)+(MetaGridDimX*MetaGridDimY*MetaBlockIdxZ);
int MetaBlock =  blockIdx.x + (blockIdx.y * gridDim.x) + (gridDim.x * gridDim.y * blockIdx.z)*MetaID;
int MetaThread =  MetaBlock * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z *blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x)+ threadIdx.x;
int blockId = blockIdx.x + (blockIdx.y * gridDim.x) + (gridDim.x * gridDim.y * blockIdx.z);
int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z *blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x)+ threadIdx.x;
int BatchVolume = (blockDim.x*gridDim.x) *(blockDim.y*gridDim.y) *(blockDim.z*gridDim.z);

        if  (threadId<BatchVolume){
            if (MetaThread<OriginalTotalVolume){
                nMem[BatchIndex*BatchVolume+threadId]=  oMem[MetaThread]  ;
            }else{
                nMem[BatchIndex*BatchVolume+threadId]=0.0;
            }
        } 
       }
extern "C" __global__
void CHWfromSegmentedNCHWfloat(const int BatchIndex,
                              const int MetaBlockIdxX,
                              const int MetaBlockIdxY,
                              const int MetaBlockIdxZ,
                              const int MetaGridDimX,
                              const int MetaGridDimY,
                              const int MetaGridDimZ,
                              const int OriginalTotalVolume,
                              float *oMem,
                              float *nMem){
  
  
  int MetaID = MetaBlockIdxX +(MetaBlockIdxY*MetaGridDimX)+(MetaGridDimX*MetaGridDimY*MetaBlockIdxZ);
  int MetaBlock =  blockIdx.x + (blockIdx.y * gridDim.x) + (gridDim.x * gridDim.y * blockIdx.z)*MetaID;
  int MetaThread =  MetaBlock * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z *blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x)+ threadIdx.x;
  int blockId = blockIdx.x + (blockIdx.y * gridDim.x) + (gridDim.x * gridDim.y * blockIdx.z);
  int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z *blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x)+ threadIdx.x;
  int BatchVolume = (blockDim.x*gridDim.x) *(blockDim.y*gridDim.y) *(blockDim.z*gridDim.z);
  
          if  (threadId<BatchVolume){
              if (MetaThread<OriginalTotalVolume){
                oMem[MetaThread]=  nMem[BatchIndex*BatchVolume+threadId]   ;
              }
          } 
}     

extern "C" __global__
void l2regularizationfloat(
    const int length,
    float *dw, //input and output
    float *w,  //input
    float *l1, //output
    float *l2, //output
    const float batch, // should be an int but just send it as a float
    const float decay1,
    const float decay2){
        int i=  (blockIdx.y*gridDim.x*blockDim.x) +(blockIdx.x*blockDim.x) + threadIdx.x;
    
    if (i<length){
    atomicAdd(l2,(w[i]*w[i]*decay2)/2.0);
    dw[i]= (dw[i]/batch) + w[i]*decay2;
    }
 
}  
extern "C" __global__
void batchregfloat(
    const int length,
    float *dw, //input and output
    const float batch) {// should be an int but just send it as a float
        int i=  (blockIdx.y*gridDim.x*blockDim.x) +(blockIdx.x*blockDim.x) + threadIdx.x;
    if (i<length){
   
    dw[i]/=batch;
    }
 
}  
extern "C" __global__
void l1l2regularizationfloat(
    const int length,
    float *dw, //input and output
    float *w,  //input needs to ba an array
   // int values, //number of values
    float *l1, //output set to zero
    float *l2, //output set to zero
    const float batch, // should be an int but just send it as a float
    const float decay1, //input
    const float decay2){ //input
 
int i=  (blockIdx.y*gridDim.x*blockDim.x) +(blockIdx.x*blockDim.x) + threadIdx.x;
   int decay =decay1 ;
    if (i<length){
        
        if (dw[i]<0){
            decay=-decay;
        }

        atomicAdd(l1,w[i]*decay); 
        atomicAdd(l2,(w[i]*w[i]*decay2)/2.0);
        dw[i]= (dw[i]/batch) + (w[i]*decay2) +decay1;
     }

}  


extern "C" __global__
void forwardParametricfloat(const int length, const int alphalength ,float *x,float *y,  float *alpha){
   int xsize = gridDim.x*blockDim.x;
   int i= blockIdx.x*blockDim.x+threadIdx.x;
   int j = xsize*blockIdx.y+i;

  
if (j<length){
    if (i<alphalength){
        if (x[j]>0.0){
            y[j]=x[j];
        }else{
            y[j]=alpha[i]*x[j];
        }


    }
    
    
}

}

//NHCW, NCWH only matters on the batch channel so for this to work alpha and dalpha are going to have to be the size of 
// HCW.  
extern "C" __global__  
void backwardParametricfloat(const int length, const int alphalength ,float *x, float *dx,float *dy,  float *alpha, float *dalpha){

    int xsize = gridDim.x*blockDim.x;
    int i= blockIdx.x*blockDim.x+threadIdx.x;
    int j = xsize*blockIdx.y+i;
 
if (j<length){
    if (i<alphalength){
    if (x[j]>0.0){
        dx[j]=dy[j];
    }else{
        dx[j]=alpha[i]*dy[j];
        atomicAdd(&dalpha[i],x[j]*dy[j]);
  
    }   
 
}
}
}
extern "C" __global__
void forwardleakyfloat(const int length,float *x,float *y, const float alpha){

int i=  (blockIdx.y*gridDim.x*blockDim.x) +(blockIdx.x*blockDim.x) + threadIdx.x;
if (i<length){
    if (x[i]>0.0){
        y[i]=x[i];
    }else{
        y[i]=alpha*x[i];
    }
    
}

}   

/*
extern "C" __global__
void concatforwardleakyfloatNCHW(const int length, const int batch, const int xAlength, const int xBlength, const int ylength, float *xA, float xB, float *y, const float alpha){

int i=  (blockIdx.y*gridDim.x*blockDim.x) +(blockIdx.x*blockDim.x) + threadIdx.x;
if (i<xAlength){
    if (xA[i*batch]>0.0){
        y[i]=xA[i*batch];
    }else{
        y[i*batch*(ylength)]=alpha*xA[i*batch];
    }
    
}
if (i<xBlength){
    if (xA[i*batch]>0.0){
        y[i]=xB[i*batch];
    }else{
        y[i*batch*(ylength+xAlength)]=alpha*xB[i*batch];
    }
    
}
}   
*/

extern "C" __global__
void backwardleakyfloat(const int length,float *x, float *dx,float *dy, const float alpha){
int i=  (blockIdx.y*gridDim.x*blockDim.x) +(blockIdx.x*blockDim.x) + threadIdx.x;
if (i<length){
    if (x[i]>0.0){
        dx[i]=dy[i];
    }else{
        dx[i]=alpha*dy[i];
    }
    
}

}  
