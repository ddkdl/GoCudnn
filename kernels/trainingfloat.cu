
#include <hip/hip_runtime.h>
extern "C" __global__
void adagradfloat(float *weights, //weights input and output
                  float *gsum, //storage
                  float *dw, //input and will have to set to zero
                  float rate, //input
                  float eps){ //input
                
 
    int section = blockIdx.x;
    int index = threadIdx.x;
    int cell = section*blockDim.x +index;
    gsum[cell]= gsum[cell]+(dw[cell]*dw[cell]);
    weights[cell]= -(rate*dw[cell])/(sqrtf(gsum[cell])+eps);
}


extern "C" __global__
void adamfloat(
          float *w,
          float *gsum,
          float *xsum,
          float *dw,
          float beta1,
          float beta2,
          float eps,
          float counter){

    int section = blockIdx.x;
    int index = threadIdx.x;
    int cell = section*blockDim.x +index;


    gsum[cell]=beta1*gsum[cell] +(1.0-beta1)*dw[cell];
    float gsumt = 0;
    gsumt = gsum[cell]/(1.0- powf(beta1,counter));
    xsum[cell]= (beta2*xsum[cell])+((1.0 -beta2)*dw[cell]*dw[cell]);
    float xsumt =0;
    xsumt = xsum[cell]/(1.0 - powf(beta2,counter));
    //float hw = w[cell];
    w[cell] +=  -(eps*gsumt)/(sqrtf(xsumt)+eps);      
}


extern "C" __global__
void adadeltafloat(
                    float *weights, //weights input and output
                    float *gsum, //storage
                    float *xsum, //storage
                    float *dw, //input and will have to set to zero
                    const float rate, //input
                    const float eps){



            int section = blockIdx.x;
            int index = threadIdx.x;
            int cell = section*blockDim.x +index;

gsum[cell]= gsum[cell]+(dw[cell]*dw[cell]);
weights[cell]= -(rate*dw[cell])/(sqrtf(gsum[cell])+eps);
dw[cell]=0.0;
}

extern "C" __global__
void l1regularizationfloat(
    float *dw, //input and output
    float *w,  //input
 //   int values, //number of values
    float *l1, //output
    float *l2, //output
    float batch, // should be an int but just send it as a float
    float decay1,
    const float decay2){

    int section = blockIdx.x;
    int index = threadIdx.x;
    int cell = section*blockDim.x+index;
    float decay = decay1;
    if (dw[cell]<0){
        decay=-decay;
    }
 //   __syncthreads();
    atomicAdd(l1,w[cell]*decay);
  //  __syncthreads();
    dw[cell]= (dw[cell]/batch) +decay;
    
}  

extern "C" __global__
void l2regularizationfloat(
    float *dw, //input and output
    float *w,  //input
    //int values, //number of values
    float *l1, //output
    float *l2, //output
    const float batch, // should be an int but just send it as a float
    const float decay1,
    const float decay2){

    int section = blockIdx.x;
    int index = threadIdx.x;
    int cell = section*blockDim.x+index;
//    __syncthreads();
    atomicAdd(l2,(w[cell]*w[cell]*decay2)/2.0);
 //   __syncthreads();
    dw[cell]= (dw[cell]/batch) + w[cell]*decay2;

    
}  

extern "C" __global__
void l1l2regularizationfloat(
    float *dw, //input and output
    float *w,  //input needs to ba an array
   // int values, //number of values
    float *l1, //output set to zero
    float *l2, //output set to zero
   const float batch, // should be an int but just send it as a float
   const float decay1, //input
   const float decay2 ){ //input
    int section = blockIdx.x;
    int index = threadIdx.x;
    int cell = section*blockDim.x+index;
    float decay = decay1;
    if (dw[cell]<0){
        decay=-decay;
    }
 //   __syncthreads();
    atomicAdd(l1,w[cell]*decay);
  //  __syncthreads();

    atomicAdd(l2,(w[cell]*w[cell]*decay2)/2.0);
   // __syncthreads();
    dw[cell]= (dw[cell]/batch) + (w[cell]*decay2) +decay1;
    
}  
