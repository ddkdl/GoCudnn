
#include <hip/hip_runtime.h>
extern "C" __global__
void adagradfloat(const int length,
                  float *weights, //weights input and output
                  float *dw, //input and will have to set to zero
                  float *gsum, //storage
                  const float rate, //input
                  const float eps){ //input
                
 
    int section = blockIdx.x;
    int index = threadIdx.x;
    int cell = section*blockDim.x +index;
    if (cell<length){
        int holder = gsum[cell];
        gsum[cell]= holder +(dw[cell]*dw[cell]);
        weights[cell]= -(rate*dw[cell])/(sqrtf(gsum[cell])+eps);
        dw[cell]=0.0;
    }  

}


extern "C" __global__
void adamfloat(const int length,
               float *w,
               float *gsum,
               float *xsum,
               float *dw,
               const float beta1,
               const float beta2,
               const float eps,
               const float counter){

    
    int cell = (blockIdx.y*gridDim.x*blockDim.x) +
    (blockIdx.x*blockDim.x) + 
    threadIdx.x;

if (cell<length){
    float ghold=gsum[cell];
    gsum[cell]=beta1*ghold +(1.0-beta1)*dw[cell];
    float gsumt = 0;
    gsumt = gsum[cell]/(1.0- powf(beta1,counter));
    float xhold=xsum[cell];
    xsum[cell]= (beta2*xhold)+((1.0 -beta2)*dw[cell]*dw[cell]);
    float xsumt =0;
    xsumt = xsum[cell]/(1.0 - powf(beta2,counter));
    //float hw = w[cell];
    float wcellhold = w[cell];
    w[cell] = wcellhold -(eps*gsumt)/(sqrtf(xsumt)+eps);  
    __syncthreads();
    dw[cell]=0.0;

}
    
}


extern "C" __global__
void adadeltafloat( const int length,
                    float *weights, //weights input and output
                    float *gsum, //storage
                    float *xsum, //storage
                    float *dw, //input and will have to set to zero
                    const float rate, //input
                    const float eps){



            int section = blockIdx.x;
            int index = threadIdx.x;
            int cell = section*blockDim.x +index;
if(cell<length){
    gsum[cell]= gsum[cell]+(dw[cell]*dw[cell]);
    weights[cell]= -(rate*dw[cell])/(sqrtf(gsum[cell])+eps);
    dw[cell]=0.0;


}

}

extern "C" __global__
void l1regularizationfloat(const int length,
                           float *dw, //input and output
                           float *w,  //input
                           float *l1, //output
                           float *l2, //output
                           const float batch, // should be an int but just send it as a float
                           const float decay1,
                           const float decay2){

        int section = blockIdx.x;
        int index = threadIdx.x;
        int cell = section*blockDim.x+index;
        float decay = decay1;
        if (cell<length){
            if (dw[cell]<0){
                decay=-decay;
            }
            atomicAdd(l1,w[cell]*decay);
            dw[cell]= (dw[cell]/batch) +decay;


       }

    
}  

extern "C" __global__
void l2regularizationfloat(
    const int length,
    float *dw, //input and output
    float *w,  //input
    float *l1, //output
    float *l2, //output
    const float batch, // should be an int but just send it as a float
    const float decay1,
    const float decay2){
    int section = blockIdx.x;
    int index = threadIdx.x;
    int cell = section*blockDim.x+index;
    if (cell<length){
    atomicAdd(l2,(w[cell]*w[cell]*decay2)/2.0);
    dw[cell]= (dw[cell]/batch) + w[cell]*decay2;
    }
 
}  

extern "C" __global__
void l1l2regularizationfloat(
    const int length,
    float *dw, //input and output
    float *w,  //input needs to ba an array
   // int values, //number of values
    float *l1, //output set to zero
    float *l2, //output set to zero
    const float batch, // should be an int but just send it as a float
    const float decay1, //input
    const float decay2){ //input
    int section = blockIdx.x;
    int index = threadIdx.x;
    int cell = section*blockDim.x+index;
    float decay = decay1;
    
    if (cell<length){
        
        if (dw[cell]<0){
            decay=-decay;
        }

        atomicAdd(l1,w[cell]*decay); 
        atomicAdd(l2,(w[cell]*w[cell]*decay2)/2.0);
        dw[cell]= (dw[cell]/batch) + (w[cell]*decay2) +decay1;
     }

}  


extern "C"  __global__ 
void simpleadds(
    const int length,
    float *dw, //input and output
    float *w,  //input
    float *gsum, // should be an int but just send it as a float
    float *xsum){

    int section = blockIdx.x;
    int index = threadIdx.x;
    int cell = section*blockDim.x+index;
if (cell<length){
    gsum[cell]=dw[cell]+w[cell];
    xsum[cell]=dw[cell]+w[cell];
}
}

extern "C" __global__
void copyto(const int length,float *dest,float *src){

int i=  (blockIdx.y*gridDim.x*blockDim.x) +(blockIdx.x*blockDim.x) + threadIdx.x;
if (i<length){
    dest[i]=src[i];
}

}
    
  
