#include "hip/hip_runtime.h"
extern "C" __global__
void adagradfloat(float *weights, //weights input and output
                  float *gsum, //storage
                  float *dw, //input and will have to set to zero
                  float *loss1, //output
                  float *loss2, //output
                  float rate, //input
                  float decay1,//input
                  float decay2,//input
                  int batch, 
                  float eps){
                
 
    int section = blockIdx.x;
    int index = threadIdx.x;
    int cell = section*blockDim.x +index;
    int decaya
    if (weight[cell]<0.0){
        decaya=-decay1
    }
    gsum[cell]= gsum[cell]+(dw[cell]*dw[cell]);
    weights[cell]= -(rate*dw[cell])/(sqrtf(gsum[cell])+eps);
}


void adadeltafloat(
                    float *weights, //weights input and output
                    float *gsum, //storage
                    float *xsum, //storage
                    float *dw, //input and will have to set to zero
                    float *loss1, //output
                    float *loss2, //output
                    const float rate, //input
                    const float decay1,//input
                    const float decay2,//input from cpu
                    const int batch, //input from cpu 
                    const float eps){



            int section = blockIdx.x;
            int index = threadIdx.x;
            int cell = section*blockDim.x +index;

            if weights[cell]<0.0{
            decay1=-decay1;
            }
decay2=weights[cell]*decay2;
dw[cell]=(dw[cell]/(float)batch)+decay+decay2;
gsum[cell]= gsum[cell]+(dw[cell]*dw[cell]);
weights[cell]= -(rate*dw[cell])/(sqrtf(gsum[cell])+eps);
dw[cell]=0.0;

}

extern "C" __global__
void l1regularizationfloat(
    float *dw, //input and output
    float *w  //input
    int values, //number of values
    float *l1, //output
    float batch, // should be an int but just send it as a float
    float decay1,
){
    int section = blockIdx.x;
    int index = threadIdx.x;
    int cell = section*blockDim.x+index;
    float decay = decay1;
    if (dw[cell]<0){
        decay=-decay;
    }
    atomicAdd(&l1,w[cell]*decay);
    dw[cell]= (dw[cell]/batch) +decay;
    
}  
extern "C" __global__
void l2regularizationfloat(
    float *dw, //input and output
    float *w  //input
    //int values, //number of values
    float *l2, //output
    const float batch, // should be an int but just send it as a float
    const float decay2,
){
    int section = blockIdx.x;
    int index = threadIdx.x;
    int cell = section*blockDim.x+index;
  

    atomicAdd(&l2,(w[cell]*w[cell]*decay2)/2.0);
    dw[cell]= (dw[cell]/batch) + w[cell]*decay2;
    
}  
extern "C" __global__
void l1l2regularizationfloat(
    float *dw, //input and output
    float *w  //input needs to ba an array
   // int values, //number of values
    float *l1, //output set to zero
    float *l2, //output set to zero
   const float batch, // should be an int but just send it as a float
   const float decay1, //input
   const float decay2, //input
){
    int section = blockIdx.x;
    int index = threadIdx.x;
    int cell = section*blockDim.x+index;
    float decay = decay1;
    if (dw[cell]<0){
        decay=-decay;
    }

    atomicAdd(&l1,w[cell]*decay);
    atomicAdd(&l2,(w[cell]*w[cell]*decay2)/2.0);
    dw[cell]= (dw[cell]/batch) + (w[cell]*decay2) +decay1;
    
}  

