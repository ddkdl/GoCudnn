#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <hip/hip_fp16.h>

#define CUDA_GRID_LOOP_X(i, n)                                 \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
         i += blockDim.x * gridDim.x)

#define CUDA_GRID_AXIS_LOOP(i, n, axis)                                 \
    for (int i = blockIdx.axis * blockDim.axis + threadIdx.axis; i < n; \
         i += blockDim.axis * gridDim.axis)




//forwardPrelu does the forward Prelu

    
//backwardPrelu does the backprop of the parametric float


/* 
Leaky functions
*/














//MakePlanarImageBatchesUint8 - for this to work all the each batch should have the same amount of channels and all the channels
//need to be the same size 
extern "C" __global__ void MakePlanarImageBatchesUint8(const int XThreads, //Should be channel size
                                                 const int Batches,
                                                 const int channelsperbatch,
                                                 const float *Srcs, //all the channels for everything.
                                                 float *dest)
{
    const int batchsize = XThreads*channelsperbatch;
    for (int i = 0;i<Batches;i++)
    {
        for (int j = 0;j<channelsperbatch;j++)
        {
            CUDA_GRID_LOOP_X(xIdx, XThreads)
            {
               dest[(i*batchsize)+(j*XThreads)+xIdx]=Srcs[(j*XThreads)+xIdx];
            }
        }
    
    }
}